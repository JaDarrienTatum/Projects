#include "hip/hip_runtime.h"
// nvcc HW15.cu -o temp

#include <sys/time.h>
#include <stdio.h>
#include "./MyCuda.h"

#define DATA_CHUNKS (1024*1024) 
#define ENTIRE_DATA_SET (20*DATA_CHUNKS)
#define MAX_RANDOM_NUMBER 1000
#define BLOCK_SIZE 256

//Function prototypes
void setUpCudaDevices();
void allocateMemory();
void loadData();
void cleanUp();
__global__ void trigAdditionGPU(float *, float *, float *, int );

//Globals
dim3 BlockSize; //This variable will hold the Dimensions of your block
dim3 GridSize; //This variable will hold the Dimensions of your grid
float *NumbersOnGPU, *PageableNumbersOnCPU, *PageLockedNumbersOnCPU;
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A0_GPU, *B0_GPU, *C0_GPU, *A1_GPU, *B1_GPU, *C1_GPU; //GPU pointers
hipEvent_t StartEvent, StopEvent;
hipStream_t Stream0, Stream1;

//This will be the layout of the parallel space we will be using.
void setUpCudaDevices()
{
	hipEventCreate(&StartEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventCreate(&StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	hipDeviceProp_t prop;
	int whichDevice;
	
	hipGetDevice(&whichDevice);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	hipGetDeviceProperties(&prop, whichDevice);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	if(prop.deviceOverlap != 1)
	{
		printf("\n GPU will not handle overlaps so no speedup from streams");
		printf("\n Good bye.");
		exit(0);
	}
	
	hipStreamCreate(&Stream0);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipStreamCreate(&Stream1);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	BlockSize.x = BLOCK_SIZE;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	if(DATA_CHUNKS%BLOCK_SIZE != 0)
	{
		printf("\n Data chunks do not divide evenly by block size, sooo this program will not work.");
		printf("\n Good bye.");
		exit(0);
	}
	GridSize.x = DATA_CHUNKS/BLOCK_SIZE;
	GridSize.y = 1;
	GridSize.z = 1;	
}

//Sets a side memory on the GPU and CPU for our use.
void allocateMemory()
{	
	//Allocate Device (GPU) Memory
	hipMalloc(&A0_GPU,DATA_CHUNKS*sizeof(float));
	myCudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B0_GPU,DATA_CHUNKS*sizeof(float));
	myCudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C0_GPU,DATA_CHUNKS*sizeof(float));
	myCudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&A1_GPU,DATA_CHUNKS*sizeof(float));
	myCudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B1_GPU,DATA_CHUNKS*sizeof(float));
	myCudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C1_GPU,DATA_CHUNKS*sizeof(float));
	myCudaErrorCheck(__FILE__, __LINE__);
	
	//Allocate page locked Host (CPU) Memory
	hipHostAlloc(&A_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipHostAlloc(&B_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipHostAlloc(&C_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	myCudaErrorCheck(__FILE__, __LINE__);
}

void loadData()
{
	time_t t;
	srand((unsigned) time(&t));
	
	for(int i = 0; i < ENTIRE_DATA_SET; i++)
	{		
		A_CPU[i] = MAX_RANDOM_NUMBER*rand()/RAND_MAX;
		B_CPU[i] = MAX_RANDOM_NUMBER*rand()/RAND_MAX;	
	}
}

//Cleaning up memory after we are finished.
void cleanUp()
{
	hipFree(A0_GPU); 
	myCudaErrorCheck(__FILE__, __LINE__);
	hipFree(B0_GPU); 
	myCudaErrorCheck(__FILE__, __LINE__);
	hipFree(C0_GPU); 
	myCudaErrorCheck(__FILE__, __LINE__);
	hipFree(A1_GPU); 
	myCudaErrorCheck(__FILE__, __LINE__);
	hipFree(B1_GPU); 
	myCudaErrorCheck(__FILE__, __LINE__);
	hipFree(C1_GPU); 
	myCudaErrorCheck(__FILE__, __LINE__);
	
	hipHostFree(A_CPU);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipHostFree(B_CPU);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipHostFree(C_CPU);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	hipEventDestroy(StartEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventDestroy(StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	hipStreamDestroy(Stream0);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipStreamDestroy(Stream1);
	myCudaErrorCheck(__FILE__, __LINE__);
}

__global__ void trigAdditionGPU(float *a, float *b, float *c, int n)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(id < n)
	{
		c[id] = sin(a[id]) + cos(b[id]);
	}
}

int main()
{
	float timeEvent;
	
	setUpCudaDevices();
	allocateMemory();
	loadData();
	
	hipEventRecord(StartEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	for(int i = 0; i < ENTIRE_DATA_SET; i += DATA_CHUNKS*2)
	{
		//******************************************
		hipMemcpyAsync(A0_GPU, A_CPU + i, DATA_CHUNKS*sizeof(float), hipMemcpyHostToDevice,Stream0);
		myCudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(B0_GPU, B_CPU + i, DATA_CHUNKS*sizeof(float), hipMemcpyHostToDevice,Stream0);
		myCudaErrorCheck(__FILE__, __LINE__);
		trigAdditionGPU<<<DATA_CHUNKS/BLOCK_SIZE,BLOCK_SIZE,0,Stream0>>>(A0_GPU, B0_GPU, C0_GPU, DATA_CHUNKS);
		hipMemcpyAsync(C_CPU + i, C0_GPU,DATA_CHUNKS*sizeof(float), hipMemcpyDeviceToHost, Stream0);
		myCudaErrorCheck(__FILE__, __LINE__);
		
		hipMemcpyAsync(A1_GPU, A_CPU + i + DATA_CHUNKS, DATA_CHUNKS*sizeof(float), hipMemcpyHostToDevice,Stream1);
		myCudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(B1_GPU, B_CPU + i + DATA_CHUNKS, DATA_CHUNKS*sizeof(float), hipMemcpyHostToDevice,Stream1);
		myCudaErrorCheck(__FILE__, __LINE__);
		trigAdditionGPU<<<DATA_CHUNKS/BLOCK_SIZE,BLOCK_SIZE,0,Stream1>>>(A1_GPU, B1_GPU, C1_GPU, DATA_CHUNKS);
		hipMemcpyAsync(C_CPU + i + DATA_CHUNKS, C1_GPU,DATA_CHUNKS*sizeof(float), hipMemcpyDeviceToHost, Stream1);
		myCudaErrorCheck(__FILE__, __LINE__);
		//******************************************
	}
	
	// Make the CPU wait until the Streams have finishd before it continues.
	hipStreamSynchronize(Stream0);
	hipStreamSynchronize(Stream1);
	
	hipEventRecord(StopEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	// Make the CPU wiat until this event finishes so the timing will be correct.
	hipEventSynchronize(StopEvent); 
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU = %3.1f milliseconds", timeEvent);
	
	
	printf("\n");
	//You're done so cleanup your mess.
	cleanUp();	
	
	return(0);
}
