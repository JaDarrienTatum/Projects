#include "hip/hip_runtime.h"
// nvcc HW14.cu -o temp

#include <sys/time.h>
#include <stdio.h>
#include "./MyCuda.h"

#define DATA_CHUNKS (1024*1024) 
#define ENTIRE_DATA_SET (20*DATA_CHUNKS)
#define MAX_RANDOM_NUMBER 1000
#define BLOCK_SIZE 256

//Function prototypes
void setUpCudaDevices();
void allocateMemory();
void loadData();
void cleanUp();
__global__ void trigAdditionGPU(float *, float *, float *, int );

//Globals
dim3 BlockSize; //This variable will hold the Dimensions of your block
dim3 GridSize; //This variable will hold the Dimensions of your grid
float *NumbersOnGPU, *PageableNumbersOnCPU, *PageLockedNumbersOnCPU;
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers
hipEvent_t StartEvent, StopEvent;

// Notice that we have to define a stream
hipStream_t Stream0;

//This will be the layout of the parallel space we will be using.
void setUpCudaDevices()
{
	hipEventCreate(&StartEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventCreate(&StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	hipDeviceProp_t prop;
	int whichDevice;
	
	hipGetDevice(&whichDevice);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	hipGetDeviceProperties(&prop, whichDevice);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	if(prop.deviceOverlap != 1)
	{
		printf("\n GPU will not handle overlaps so no speedup from streams");
		printf("\n Good bye.");
		exit(0);
	}
	
	// Notice that we have to create the stream
	hipStreamCreate(&Stream0);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	BlockSize.x = BLOCK_SIZE;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	if(DATA_CHUNKS%BLOCK_SIZE != 0)
	{
		printf("\n Data chunks do not divide evenly by block size, sooo this program will not work.");
		printf("\n Good bye.");
		exit(0);
	}
	GridSize.x = DATA_CHUNKS/BLOCK_SIZE;
	GridSize.y = 1;
	GridSize.z = 1;	
}

//Sets a side memory on the GPU and CPU for our use.
void allocateMemory()
{	
	//Allocate Device (GPU) Memory
	hipMalloc(&A_GPU,DATA_CHUNKS*sizeof(float));
	myCudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B_GPU,DATA_CHUNKS*sizeof(float));
	myCudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C_GPU,DATA_CHUNKS*sizeof(float));
	myCudaErrorCheck(__FILE__, __LINE__);
	
	// Notice that we are using host page locked memory
	//Allocate page locked Host (CPU) Memory
	hipHostAlloc(&A_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipHostAlloc(&B_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipHostAlloc(&C_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	myCudaErrorCheck(__FILE__, __LINE__);
}

void loadData()
{
	time_t t;
	srand((unsigned) time(&t));
	
	for(int i = 0; i < ENTIRE_DATA_SET; i++)
	{		
		A_CPU[i] = MAX_RANDOM_NUMBER*rand()/RAND_MAX;
		B_CPU[i] = MAX_RANDOM_NUMBER*rand()/RAND_MAX;	
	}
}

//Cleaning up memory after we are finished.
void cleanUp()
{
	hipFree(A_GPU); 
	myCudaErrorCheck(__FILE__, __LINE__);
	hipFree(B_GPU); 
	myCudaErrorCheck(__FILE__, __LINE__);
	hipFree(C_GPU); 
	myCudaErrorCheck(__FILE__, __LINE__);
	
	// Notice that we have to free this memory with hipHostFree
	hipHostFree(A_CPU);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipHostFree(B_CPU);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipHostFree(C_CPU);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	hipEventDestroy(StartEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventDestroy(StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	// Notice that we have to kill the stream.
	hipStreamDestroy(Stream0);
	myCudaErrorCheck(__FILE__, __LINE__);
}

__global__ void trigAdditionGPU(float *a, float *b, float *c, int n)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(id < n)
	{
		c[id] = sin(a[id]) + cos(b[id]);
	}
}

int main()
{
	float timeEvent;
	
	setUpCudaDevices();
	allocateMemory();
	loadData();
	
	hipEventRecord(StartEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	for(int i = 0; i < ENTIRE_DATA_SET; i += DATA_CHUNKS)
	{
		hipMemcpyAsync(A_GPU, A_CPU + i, DATA_CHUNKS*sizeof(float), hipMemcpyHostToDevice,Stream0);
		myCudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(B_GPU, B_CPU + i, DATA_CHUNKS*sizeof(float), hipMemcpyHostToDevice,Stream0);
		myCudaErrorCheck(__FILE__, __LINE__);
		trigAdditionGPU<<<DATA_CHUNKS/BLOCK_SIZE,BLOCK_SIZE,0,Stream0>>>(A_GPU, B_GPU, C_GPU, DATA_CHUNKS);
		hipMemcpyAsync(C_CPU + i, C_GPU,DATA_CHUNKS*sizeof(float), hipMemcpyDeviceToHost, Stream0);
		myCudaErrorCheck(__FILE__, __LINE__);
	}
	
	// Notice that we have make the CPU wait until the GPU has finished stream0
	hipStreamSynchronize(Stream0); 
	
	hipEventRecord(StopEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	// Make the CPU wiat until this event finishes so the timing will be correct.
	hipEventSynchronize(StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU = %3.1f milliseconds", timeEvent);
	
	
	printf("\n");
	//You're done so cleanup your mess.
	cleanUp();	
	
	return(0);
}
