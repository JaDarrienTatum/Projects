// nvcc HW13.cu -o temp

#include <sys/time.h>
#include <stdio.h>
#include "./MyCuda.h"

#define SIZE 2000000 
#define NUMBER_OF_COPIES 1000

//Function prototypes
void setUpCudaDevices();
void allocateMemory();
void cleanUp();
void copyPageableMemoryUp();
void copyPageLockedMemoryUp();
void copyPageableMemoryDown();
void copyPageLockedMemoryDown();

//Globals
float *NumbersOnGPU, *PageableNumbersOnCPU, *PageLockedNumbersOnCPU;
hipEvent_t StartEvent, StopEvent;

//This will be the layout of the parallel space we will be using.
void setUpCudaDevices()
{
	hipEventCreate(&StartEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventCreate(&StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
}

//Sets a side memory on the GPU and CPU for our use.
void allocateMemory()
{					
	//Allocate Device (GPU) Memory
	hipMalloc(&NumbersOnGPU, SIZE*sizeof(float));
	myCudaErrorCheck(__FILE__, __LINE__);

	//Allocate pageable Host (CPU) Memory
	PageableNumbersOnCPU = (float*)malloc(SIZE*sizeof(float));
	
	//Allocate page locked Host (CPU) Memory
	hipHostAlloc(&PageLockedNumbersOnCPU, SIZE*sizeof(float), hipHostMallocDefault);
	myCudaErrorCheck(__FILE__, __LINE__);
}

//Cleaning up memory after we are finished.
void cleanUp()
{
	hipFree(NumbersOnGPU); 
	myCudaErrorCheck(__FILE__, __LINE__);
	
	hipHostFree(PageLockedNumbersOnCPU);
	myCudaErrorCheck(__FILE__, __LINE__);
	
	free(PageableNumbersOnCPU); 
	
	hipEventDestroy(StartEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventDestroy(StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
}

void copyPageableMemoryUp()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		hipMemcpy(NumbersOnGPU, PageableNumbersOnCPU, SIZE*sizeof(float), hipMemcpyHostToDevice);
		myCudaErrorCheck(__FILE__, __LINE__);
	}
}

void copyPageableMemoryDown()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		hipMemcpy(PageableNumbersOnCPU, NumbersOnGPU, SIZE*sizeof(float), hipMemcpyDeviceToHost);
		myCudaErrorCheck(__FILE__, __LINE__);
	}
}

void copyPageLockedMemoryUp()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		hipMemcpy(NumbersOnGPU, PageLockedNumbersOnCPU, SIZE*sizeof(float), hipMemcpyHostToDevice);
		myCudaErrorCheck(__FILE__, __LINE__);
	}
}

void copyPageLockedMemoryDown()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		hipMemcpy(PageLockedNumbersOnCPU, NumbersOnGPU, SIZE*sizeof(float), hipMemcpyDeviceToHost);
		myCudaErrorCheck(__FILE__, __LINE__);
	}
}


int main()
{
	float timeEvent;
	
	setUpCudaDevices();
	allocateMemory();
	
	hipEventRecord(StartEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	copyPageableMemoryUp();
	hipEventRecord(StopEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using pageable memory up = %3.1f milliseconds", timeEvent);
	
	hipEventRecord(StartEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	copyPageLockedMemoryUp();
	hipEventRecord(StopEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using page locked memory up = %3.1f milliseconds", timeEvent);
	
	hipEventRecord(StartEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	copyPageableMemoryDown();
	hipEventRecord(StopEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using pageable memory down = %3.1f milliseconds", timeEvent);
	
	hipEventRecord(StartEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	copyPageLockedMemoryDown();
	hipEventRecord(StopEvent, 0);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	myCudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using page locked memory down = %3.1f milliseconds", timeEvent);
	
	printf("\n");
	//You're done so cleanup your mess.
	cleanUp();	
	
	return(0);
}
